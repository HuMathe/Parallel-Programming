#include "hip/hip_runtime.h"
#include "gui.h"
#include "physics.h"
#include "signature.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

#define whereis(node, x, y) (       \
  (node.lx + node.rx > x + x)       \
    ? (node.uy + node.dy > y + y ? 0 : 2) \
    : (node.uy + node.dy > y + y ? 1 : 3))


// extern __shared__ particle2d_stat_t stars[];

// extern __shared__ double position[];

__device__ void get_g(double x0, double y0, double x1, double y1, double m1, 
  double &ansx, double &ansy, double min_r, double eps) {

  double dx, dy;
  double norm;
  dx = x1 - x0;
  dy = y1 - y0;
  norm = sqrt(dx * dx + dy * dy);
  if(norm > min_r) {
    ansx += dx * m1 / (norm * norm + eps) / norm;
    ansy += dy * m1 / (norm * norm + eps) / norm;
  }
}

__device__ void collision(double x0, double y0, double m0, double &vx0, double &vy0, 
  double x1, double y1, double m1, double vx1, double vy1,
  double min_r, double eps) {

  double dx, dy;
  double norm, coeff;
  dx = x0 - x1;
  dy = y0 - y1;
  norm = sqrt(dx * dx + dy * dy);
  if(vx0 * vx0 + vy0 * vy0 > 25e8) {
    vx0 = vx0 / (sqrt(vx0 * vx0 + vy0 * vy0)) * 3e4;
    vy0 = vy0 / (sqrt(vx0 * vx0 + vy0 * vy0)) * 3e4;
  }
  if(norm <= min_r) {
    coeff = 2.0 * m1 / (m0 + m1) / (norm + eps) / (norm + eps);
    coeff *= (vx0 - vx1) * (x0 - x1) + (vy0 - vy1) * (y0 - y1);
    vx0 -= coeff * (x0 - x1);
    vy0 -= coeff * (y0 - y1);
  }
}

__global__ void direct_simulation_step(particle2d_stat_t *star_stat, 
  vec2d *force, int N, const universal_const C,
  quad_tree_node_stat *ta, int use_qtree) {
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  particle2d_stat_t *stars = star_stat;
  
  if(i >= N) return ;

  
  force[i].x = 0;
  force[i].y = 0;
  
  if(use_qtree) {
    int cur = 0, next_cur;
    while(cur != -1) {
      for(int c = 0; c < 4; c++) {
        if(c == whereis(ta[cur], stars[i].dx, stars[i].dy)) {
          next_cur = ta[cur].ch[c];
          if(next_cur == -1) {
            get_g(stars[i].dx, stars[i].dy, 
              ta[cur].mx, ta[cur].my, ta[cur].m, 
              force[i].x, force[i].y, C.min_r, C.eps);

            collision(stars[i].dx, stars[i].dy, stars[i].m, stars[i].vx, stars[i].vy,
              ta[cur].mx, ta[cur].my, ta[cur].m, ta[cur].vx, ta[cur].vy, C.min_r, C.eps);

          }
        } else if(ta[cur].ch[c] != -1) {
          get_g(stars[i].dx, stars[i].dy, 
            ta[ta[cur].ch[c]].mx, ta[ta[cur].ch[c]].my, ta[ta[cur].ch[c]].m,
            force[i].x, force[i].y, C.min_r, C.eps);

          collision(stars[i].dx, stars[i].dy, stars[i].m, stars[i].vx, stars[i].vy,
            ta[ta[cur].ch[c]].mx, ta[ta[cur].ch[c]].my, ta[ta[cur].ch[c]].m,
            ta[ta[cur].ch[c]].vx, ta[ta[cur].ch[c]].vy, C.min_r, C.eps);

        }
      }
      cur = next_cur;
    }
  } else {
    for(int j = 0; j < N; j++) {
      if(j == i) continue;
      get_g(stars[i].dx, stars[i].dy, 
        stars[j].dx, stars[j].dy, stars[j].m,
        force[i].x, force[i].y, C.min_r, C.eps);

      collision(stars[i].dx, stars[i].dy, stars[i].m, stars[i].vx, stars[i].vy,
        stars[j].dx, stars[j].dy, stars[j].m, stars[j].vx, stars[j].vy,
        C.min_r, C.eps);
    }
  }
  force[i].x *= stars[i].m * C.G;
  force[i].y *= stars[i].m * C.G;
}

__host__ void offload2GPU(std::vector<particle2d> &stars, const universal_const C, quad_tree &tree, int use_qtree, int threadsPerBlock = 128) {


    particle2d_stat_t   *particle_stat, *d_particle_stat;
    vec2d               *forces,        *d_forces;
    quad_tree_node_stat *tree_array,    *d_tree_array;
    int tree_size = 0;

    particle_stat = new particle2d_stat_t[stars.size()];
    forces        = new vec2d[stars.size()];
    if(use_qtree) {
      tree.serialize(&tree_array, tree_size);
    }

    size_t mem_size_p = sizeof(particle2d_stat_t)   * stars.size();
    size_t mem_size_f = sizeof(vec2d)               * stars.size();
    size_t mem_size_t = sizeof(quad_tree_node_stat) * tree_size;
    
    
    hipMalloc(&d_particle_stat, mem_size_p);
    hipMalloc(&d_forces, mem_size_f);
    if(mem_size_t) hipMalloc(&d_tree_array, mem_size_t);

    for(int i = 0; i < stars.size(); i++) 
      particle_stat[i] = stars[i].stat();
    
    hipMemcpy(d_particle_stat, particle_stat, mem_size_p,
      hipMemcpyHostToDevice);
    hipMemset(d_forces, 0, mem_size_f);
    if(use_qtree) hipMemcpy(d_tree_array, tree_array, mem_size_t, 
      hipMemcpyHostToDevice);
    
    
    // offload calculation to cuda
    direct_simulation_step<<<(stars.size() - 1) / threadsPerBlock + 1, threadsPerBlock>>>(
      d_particle_stat, d_forces, stars.size(), C, d_tree_array, use_qtree);
    hipDeviceSynchronize();
    

    // load result from cuda
    hipMemcpy(forces, d_forces, mem_size_f, 
      hipMemcpyDeviceToHost);
    hipMemcpy(particle_stat, d_particle_stat, mem_size_p,
      hipMemcpyDeviceToHost);


    for(int i = 0; i < stars.size(); i++) {
      stars[i].velocity() = vec2d(particle_stat[i].vx, particle_stat[i].vy);
      stars[i].apply_constant_force(forces[i]);
    }

    // release cuda resources
    hipFree(d_particle_stat);
    hipFree(d_forces);

    // release memory
    delete[] particle_stat;
    delete[] forces;
    
}


class cuda_direct_simulation
  : public gravity_system2d {
public:
  int n_thrds;
  int blocksPerGrid, threadsPerBlock;

  cuda_direct_simulation(int l, int r, int u, int d, int use_qtree)
    : gravity_system2d(l, r, u, d, use_qtree) {
      blocksPerGrid = (bodys.size() - 1) / 128 + 1, threadsPerBlock = 128;
    }
  
  // custom parallelization
  void simulation_step() override {
    offload2GPU(bodys, this->export_env(), this->tree, use_quad_tree, 32);
  }
  
};

int main(int argc, char **argv) {
  srand(time(NULL));
  int N, n_iter;
  if(argc == 1) {
    N = 200, n_iter = 1000;
  } else {
    N = atoi(argv[1]), n_iter = atoi(argv[2]);
  }
  gui_init(&argc, argv, "cuda", 4000, 4000);
  cuda_direct_simulation s(0, 4000, 0, 4000, argc==4);

  // s.bodys.emplace_back(2000, 0, 0, 1e17);
  for(int i = 0; i < N; i++) {
    s.bodys.push_back(particle2d::random(0, 4000, 0, 4000, 1000));
  }

  for(int it = 0; it < n_iter; it++) {
    s.step();
    visualize(s.bodys);
  }

  print_info("CUDA", s.use_quad_tree);

  return 0;
}