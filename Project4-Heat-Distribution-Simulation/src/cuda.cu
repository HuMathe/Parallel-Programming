#include "hip/hip_runtime.h"
#include "heat_system.h"
#include "signature.h"
#include "gui.h"

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel_jacobi_step(double *current_state, int num_row, int num_col, double *result_buffer)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int i = idx / num_col, j = idx % num_col;
  if(idx >= num_col * num_row) return ;
  if(i == 0 || i == num_row - 1 || j == 0 || j == num_col - 1)
    result_buffer[idx] = 0.2;
  else
    result_buffer[idx] = 0.25 * (current_state[idx + 1] + current_state[idx - 1]
      + current_state[idx + num_col] + current_state[idx - num_col]);
}

__global__ void set_value(double *buffer, int *indices, int size, double value)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
  {
    buffer[indices[idx]] = value;
  }
}

class cuda_simulation
  : public grid_system
{
public:
  int ThreadPerBlock;
  double * device_state, * device_buffer;
  int * device_fire_indices, fire_size;

public:
  void init()
  {
    device_fire_indices = nullptr; // there is no fire orignally
    fire_size = 0;

    if(size == 0)
    {
      device_state = nullptr;
      device_buffer = nullptr;
      return ;
    }

    size_t buffer_size = sizeof(double) * size;
    hipMalloc(&device_state, buffer_size);
    hipMalloc(&device_buffer, buffer_size);
  }

  cuda_simulation(): grid_system() { init(); }
  cuda_simulation(int num_row, int num_col, int thread_per_block)
    : grid_system(num_row, num_col), ThreadPerBlock(thread_per_block)
  {  init(); }

  ~cuda_simulation()
  {
    if(size == 0) 
      return ;

    hipFree(device_fire_indices);
    hipFree(device_state);
    hipFree(device_buffer);
  }

  void update_state() override
  {
    kernel_jacobi_step<<<(size - 1) / ThreadPerBlock + 1, ThreadPerBlock>>>(
      device_state, num_row, num_col, device_buffer
    );
    hipDeviceSynchronize(); 
  }

  void sync_state() override
  {
    hipMemcpy(data, device_state, sizeof(double) * size, hipMemcpyDeviceToHost); 
  }

  void init_shared_memory() override
  {
    hipMemcpy(device_state, data, sizeof(double) * size, hipMemcpyHostToDevice);

    int *indices = new int[this->fire.size()];
    for(index_iterator it = this->fire.begin(); it != this->fire.end(); it++)
    {
      indices[fire_size++] = it->first * num_col + it->second;
    }
    hipMalloc(&device_fire_indices, sizeof(int) * fire_size);
    hipMemcpy(device_fire_indices, indices, sizeof(int) * fire_size, hipMemcpyHostToDevice);
    delete[] indices;
  }

  void switch_buffer() override
  {
    set_value<<<(fire_size - 1) / ThreadPerBlock + 1, ThreadPerBlock>>>(
      device_buffer, device_fire_indices, fire_size, 1.0
    );
    hipDeviceSynchronize();

    std::swap(device_state, device_buffer);
  }

};

int num_row, num_col, thread_per_block, num_iter;

int main(int argc, char **argv)
{
  if(argc == 4)
  {
    num_row = num_col = atoi(argv[1]);
    num_iter = atoi(argv[2]);
    thread_per_block = atoi(argv[3]);
  }else if(argc == 5)
  {
    num_row = atoi(argv[1]);
    num_col = atoi(argv[2]);
    num_iter = atoi(argv[3]);
    thread_per_block = atoi(argv[4]);
  }else{
    num_row = num_col = 800;
    num_iter = 100;
    thread_per_block = 256;
  }

  gui_init(&argc, argv, "CUDA", 800, 800);
  cuda_simulation heat_state(num_row, num_col, thread_per_block);

  heat_state.init_temperature(FireType::single_fire_circular);

  for(int i = 0; i < num_iter; i++)
  {
    heat_state.step();
    visualize(&heat_state, 800, 800);
  }

  print_info("CUDA");
  return 0;
}